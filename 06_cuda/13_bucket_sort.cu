
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>

__global__ void initializeBucket(int *bucket, int value) {
  bucket[threadIdx.x] = value;
}

__global__ void countKeysInBuckets(int *keys, int *bucket) {
  atomicAdd(&bucket[keys[threadIdx.x]], 1);
}

__global__ void scanBuckets(int *bucket, int *offset, int range) {
  int i = threadIdx.x;
  offset[i+1] = bucket[i];
  for(int j=1; j<range; j<<=1) {
    __syncthreads();
    int temp = (i >= j) ? offset[i] + offset[i-j] : offset[i];
    __syncthreads();
    offset[i] = temp;
  }
}

__global__ void sortKeysWithBuckets(int *keys, int *bucket, int *offset) {
  int i = threadIdx.x;
  int j = blockIdx.x;
  if (i < bucket[j]) keys[i + offset[j]] = j;
}

int main() {
  int n = 50, range = 5;
  int *keys, *bucket, *offset;

  hipMallocManaged(&keys, n * sizeof(int));
  hipMallocManaged(&bucket, range * sizeof(int));
  hipMallocManaged(&offset, (range + 1) * sizeof(int));

  for (int i = 0; i < n; i++) {
    keys[i] = rand() % range;
    printf("%d ", keys[i]);
  }
  printf("\n");

  initializeBucket<<<1, range>>>(bucket, 0);
  hipDeviceSynchronize();
  
  countKeysInBuckets<<<1, n>>>(keys, bucket);
  hipDeviceSynchronize();
  
  scanBuckets<<<1, range>>>(bucket, offset, range);
  hipDeviceSynchronize();
  
  sortKeysWithBuckets<<<range, n>>>(keys, bucket, offset);
  hipDeviceSynchronize();

  for (int i = 0; i < n; i++) {
    printf("%d ", keys[i]);
  }
  printf("\n");

  hipFree(keys);
  hipFree(bucket);
  hipFree(offset);

  return 0;
}

